
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cassert>
#define MAXN 2000
#define MULSIDE 16 // each block has size SIDE x SIDE

// function for debugging.
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void game_of_life_iter(char* A, char* B, size_t pitcha, size_t pitchb, int N){
    int localx = threadIdx.x;
    int localy = threadIdx.y;
    int globalx = localx + blockIdx.x * MULSIDE;
    int globaly = localy + blockIdx.y * MULSIDE;
 
    char alive = 0;
    char self = A[globalx * pitcha + globaly];
    alive += (globalx > 0 && globaly > 0) ? A[(globalx-1) * pitcha + (globaly-1)] : 0;
    alive += (globalx > 0) ? A[(globalx-1) * pitcha + globaly] : 0;
    alive += (globalx > 0 && globaly < N-1) ? A[(globalx-1) * pitcha + (globaly+1)] : 0;

    alive += (globaly > 0) ? A[globalx * pitcha + (globaly-1)] : 0;
    alive += (globaly < N-1) ? A[globalx * pitcha + (globaly+1)] : 0;


    alive += (globalx < N-1 && globaly > 0) ? A[(globalx+1) * pitcha + (globaly-1)] : 0;
    alive += (globalx < N-1) ? A[(globalx+1) * pitcha + globaly] : 0;
    alive += (globalx < N-1 && globaly < N-1) ? A[(globalx+1) * pitcha + (globaly+1)] : 0;
    
    if (self && (alive < 2 || alive > 3)){
        B[globalx * pitchb + globaly] = 0;
    }
    else if (!self && alive == 3){
        B[globalx * pitchb + globaly] = 1;
    }
    else{
        B[globalx * pitchb + globaly] = self;
    }
}
void copyto(char* dst, char* src, size_t pitch){
    gpuErrchk(hipMemcpy2D((void*)dst, pitch, (void *)src, MAXN, MAXN*sizeof(char), MAXN, hipMemcpyHostToDevice));
}
void copyback(char* dst, char* src, size_t pitch){
    gpuErrchk(hipMemcpy2D((void*)dst, MAXN, (void *)src, pitch, MAXN*sizeof(char), MAXN, hipMemcpyDeviceToHost));
} 
void cuClear(char* dst, size_t pitch){
    gpuErrchk(hipMemset2D((void*)dst, pitch, 0, MAXN*sizeof(char), MAXN));
}
void print_matrix(int N, char A[]) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++)
            printf("%d", A[i*MAXN + j]);
        printf("\n");
    }
}
int divCeil(int a, int b){
    int c = a / b;
    if (c * b < a){
        c++;
    }
    return c;
}
char A[MAXN*MAXN];
int main(int argc, char** argv)
{
    char digits[MAXN];
    int N, M, s;
    s = scanf("%d %d", &N, &M);
    for(int i = 0; i < N; i++){
        s = scanf("%s", digits);
        assert(s>0);
        for(int j = 0; j < N; j++){
            A[i*MAXN + j] = digits[j]=='0' ? 0 : 1;
        }
    }

    size_t pitch[2];
    char *devA[2];
    gpuErrchk(hipMallocPitch(&devA[0], &pitch[0], MAXN*sizeof(char), MAXN));
    gpuErrchk(hipMallocPitch(&devA[1], &pitch[1], MAXN*sizeof(char), MAXN));
    copyto(devA[0], (char*)A, pitch[0]);

    for (int i = 0; i < M; i++){
        int x = i%2;
        int BLOCKS = divCeil(N, MULSIDE);
        game_of_life_iter <<< dim3(BLOCKS,BLOCKS), dim3(MULSIDE,MULSIDE) >>> (devA[x], devA[!x], pitch[x], pitch[!x], N);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }
    copyback((char*)A, devA[M%2], pitch[M%2]);
    print_matrix(N, A);


    return 0;	
}
